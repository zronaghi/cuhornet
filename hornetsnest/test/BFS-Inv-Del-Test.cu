/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BFS-Reverse/BFS-Reverse.cuh"
#include <StandardAPI.hpp>
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <BufferPool.cuh>

int exec(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornets_nest;
    using vid_t = int;
    using dst_t = int;

    BufferPool pool;

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    // graph::GraphStd<vid_t, eoff_t> graph;
    graph::GraphStd<vid_t, eoff_t> graph(DIRECTED | ENABLE_INGOING);
    CommandLineParam cmd(graph, argc, argv,false);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());


    HornetInit hornet_init_inverse(graph.nV(), graph.nE(),
                                   graph.csr_in_offsets(),
                                   graph.csr_in_edges());


    std::vector<wgt0_t> edge_meta_0(graph.nE(), 0);
    hornet_init.insertEdgeData(edge_meta_0.data());
    hornet_init_inverse.insertEdgeData(edge_meta_0.data());



    HornetGraph hornet_graph_inv(hornet_init_inverse);


        // std::cout << "hornet_graph_inv : " << hornet_graph_inv.nV() << " " << hornet_graph_inv.nE() << std::endl;
        // std::cout << "hornet_graph     : " << hornet_graph.nV() << " " << hornet_graph.nE() << std::endl;

    // for(int i=0; i<10; i++){
    //     std::cout << graph.csr_in_offsets()[i] << " : " << graph.csr_out_offsets()[i] << std::endl;
    // }

 
	vid_t root = graph.max_out_degree_id();
	// if (argc==3)
	//   root = atoi(argv[2]);

    int numberRoots = 10;
    if (argc>=3)
      numberRoots = atoi(argv[2]);

    int alg = 0;
    if (argc>=4)
      alg = atoi(argv[3]);

    int deletion = 0;
    if (argc>=5)
      deletion = atoi(argv[4]);

    int timeSection = 0;
    if (argc>=6)
      timeSection = atoi(argv[5]);


    std::cout << "My root is " << root << std::endl;


    // rev_del_bfs.set_parameters(root);

    float totalTime = 0.0;

    Timer<DEVICE> TM;

    HornetGraph hornet_graph(hornet_init);
    ReverseDeleteBFS rev_del_bfs(hornet_graph, hornet_graph_inv, 
                                 graph.csr_out_offsets(),graph.csr_in_offsets());
    rev_del_bfs.sortHornets(hornet_graph_inv);
    rev_del_bfs.sortHornets(hornet_graph);
    rev_del_bfs.SetInverseIndices(hornet_graph_inv);            



    for (int i=0; i<numberRoots;i++){
        // HornetGraph hornet_graph(hornet_init);
        // ReverseDeleteBFS rev_del_bfs(hornet_graph, hornet_graph_inv);
        rev_del_bfs.reset();

        if(deletion!=0){
            // rev_del_bfs.sortHornets(hornet_graph_inv);
            // if(i==0){
            //     rev_del_bfs.sortHornets(hornet_graph_inv);
            //     rev_del_bfs.sortHornets(hornet_graph);
            // }

            // rev_del_bfs.SetInverseIndices(hornet_graph_inv);            
        }

        // hipProfilerStart();
        TM.start();
            rev_del_bfs.set_parameters((root+i)%graph.nV());
            if(deletion==0){
                rev_del_bfs.run(hornet_graph_inv,alg,timeSection);
            }
            else{
                rev_del_bfs.runNoDelete(hornet_graph_inv,alg,timeSection);
            }

        TM.stop();
        // printf("duration %f\n",TM.duration());
        totalTime += TM.duration();
        if (alg==1){
            timeSection+=0;
            totalTime -= 0.0000001;
        }
        // hipProfilerStop();
    }

    printf("\nReverse BFS time: %f ms\n",totalTime);

    int N=graph.nE();
    vid_t* d_temp;
    pool.allocate(&d_temp,N);
    TM.start();
    for (int i=0; i<5000; i++){
        hipMemset(d_temp,0,sizeof(vid_t)*N);
    }
    TM.stop();
    // gpu::free(d_temp);

    // TM.print("Reverse BFS");

    return 0;
}

int main(int argc, char* argv[]) {
    int ret = 0;
    // hornets_nest::gpu::initializeRMMPoolAllocation();//update initPoolSize if you know your memory requirement and memory availability in your system, if initial pool size is set to 0 (default value), RMM currently assigns half the device memory.
    {//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.

    ret = exec(argc, argv);

    }//scoping technique to make sure that hornets_nest::gpu::finalizeRMMPoolAllocation is called after freeing all RMM allocations.
    // hornets_nest::gpu::finalizeRMMPoolAllocation();

    return ret;
}

