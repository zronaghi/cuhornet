/**
 * @brief Breadth-first Search Top-Down test program
 * @file
 */
#include "Static/BetweennessCentrality/bc.cuh"
#include "Static/BetweennessCentrality/exact_bc.cuh"
#include "Static/BetweennessCentrality/approximate_bc.cuh"
#include <Graph/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include <omp.h>

#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>


using namespace std;
using namespace graph;
using namespace graph::structure_prop;
using namespace graph::parsing_prop;


#include <hipcub/hipcub.hpp> 

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/execution_policy.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>

#include <BufferPool.cuh>

using namespace timer;
using namespace hornets_nest;

void testSingle(graph::GraphStd<vid_t, eoff_t> &graph,int numRoots,bc_t *mgpuGlobalBC, BufferPool& pool){

    Timer<DEVICE> TM;

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                       graph.csr_out_edges());
    HornetGraph hornet_graph(hornet_init);

    vid_t* roots = new vid_t[graph.nV()];

    int i=0;
    for(int v=0; v<numRoots; v++){
        roots[i++]=v;
    }

    ApproximateBC abc(hornet_graph,roots,i);
    abc.reset();

    TM.start();

    abc.run();

    TM.stop();
    TM.print("SingleGPU Time");

    bc_t *sgpuGlobalBC,*diff;


    pool.allocate(&sgpuGlobalBC, hornet_graph.nV());
    // gpu::allocate(sgpuGlobalBC, graph.nV());
    hipMemset(sgpuGlobalBC,0, sizeof(bc_t)*graph.nV());
    hipMemcpy(sgpuGlobalBC,abc.getBCScores(),sizeof(bc_t)*graph.nV(), hipMemcpyDeviceToDevice);

    pool.allocate(&diff, hornet_graph.nV());
    // gpu::allocate(diff, graph.nV());
    hipMemset(diff,0, sizeof(bc_t)*graph.nV());

    thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), sgpuGlobalBC, diff, thrust::minus<bc_t>());

    bc_t diffS = thrust::reduce(thrust::device,  diff, diff+graph.nV(),0.0);
    bc_t sumS = thrust::reduce(thrust::device,  abc.getBCScores(), abc.getBCScores()+graph.nV(),0.0);

    cout << "Total BC scores (single) : " << sumS << endl;
    cout << "Total difference in sum is : " << diffS << endl;


    bc_t *deltaDiff;
    pool.allocate(&deltaDiff, hornet_graph.nV());
    // gpu::allocate(deltaDiff, graph.nV());

    thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), sgpuGlobalBC, deltaDiff, thrust::minus<bc_t>());
    bc_t sumSquareDelta = thrust::reduce(thrust::device,  deltaDiff, deltaDiff+graph.nV(),0.0);


    bc_t *cubsum, h_cubsum;
    gpu::allocate(cubsum, 1);

    // Determine temporary device storage requirements
    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, deltaDiff, cubsum, graph.nV());
    hipMalloc(&d_temp_storage,temp_storage_bytes);
    hipcub::DeviceReduce::Sum(d_temp_storage, temp_storage_bytes, deltaDiff, cubsum, graph.nV());
    hipFree(d_temp_storage);
    hipMemcpy(&h_cubsum,cubsum,sizeof(bc_t), hipMemcpyDeviceToHost);



    // gpu::free(cubsum);

    cout << "Total THRUSTsum square diff of delta : " << sumSquareDelta << endl;
    cout << "Total CUBsum square diff of delta : " << h_cubsum << endl;

    //    gpu::free(diff);

    // gpu::free(sgpuGlobalBC);
  
    delete[] roots;
}


int main(int argc, char* argv[]) {

    BufferPool pool;

    // GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv,false);
    Timer<DEVICE> TM;

    int numHardwareGPUs=16;
    int numGPUs=8;
    int testSingleFlag=0;

    int numRoots = 100;
    // int numRoots = graph.nV();

    if (argc >2)
        numRoots = atoi(argv[2]);
 
    if (argc >3)
        numGPUs = atoi(argv[3]);

    if (argc >4)
        testSingleFlag = atoi(argv[4]);

    hipSetDevice(0);

    bc_t *mgpuGlobalBC,*temp;

    pool.allocate(&mgpuGlobalBC, graph.nV());
    pool.allocate(&temp, graph.nV());
    
    // gpu::allocate(mgpuGlobalBC, graph.nV());
    // gpu::allocate(temp, graph.nV());

    hipMemset(mgpuGlobalBC,0, sizeof(bc_t)*graph.nV());

    int original_number_threads = 0;
    #pragma omp parallel
    {
        if (omp_get_thread_num() == 0)
          original_number_threads = omp_get_num_threads();
    }



    // cout << "Number of GPUs is : " << numGPUs << endl;
    
    // #pragma omp parallel
    {
        omp_set_num_threads(numGPUs);
    }

    bc_t* bcArray[numGPUs];
    TM.start();

    hipMemset(mgpuGlobalBC,0, sizeof(bc_t)*graph.nV());


    // Create a single Hornet Graph for each GPU
    #pragma omp parallel
    {
        int64_t thread_id = omp_get_thread_num();

//        hipSetDevice(thread_id%numHardwareGPUs);
        hipSetDevice(thread_id);

        HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                               graph.csr_out_edges());
        HornetGraph hornet_graph(hornet_init);

        vid_t* roots = new vid_t[graph.nV()/numGPUs+1];

        int i=0;
        // for(int v=thread_id; v<graph.nV(); v+=numGPUs){
        for(int v=thread_id; v<numRoots; v+=numGPUs){
            roots[i++]=v;
        }


        ApproximateBC abc(hornet_graph,roots,i);
        abc.reset();
        delete[] roots;

        abc.run();

        bcArray[thread_id] = abc.getBCScores();

        #pragma omp barrier

        #pragma omp master
        {
            hipSetDevice(0);

            for(int t=0; t<numGPUs;t++){
                hipMemcpy(temp,bcArray[t],sizeof(bc_t)*graph.nV(), hipMemcpyDeviceToDevice);
                thrust::transform(thrust::device,mgpuGlobalBC, mgpuGlobalBC+graph.nV(), temp, mgpuGlobalBC,
                   thrust::plus<bc_t>());
            }
        }
        #pragma omp barrier
    }

    hipSetDevice(0);
    TM.stop();
    TM.print("MultiGPU Time");


    // gpu::free(temp);

    bc_t sumM = thrust::reduce(thrust::device, mgpuGlobalBC,mgpuGlobalBC+graph.nV(),0.0);

    cout << "Total BC scores (multi )   : " << sumM << endl;

    if(testSingleFlag){
        testSingle(graph,numRoots, mgpuGlobalBC,pool);
    }

    #pragma omp parallel
    {
        omp_set_num_threads(original_number_threads);
    }

    // gpu::free(mgpuGlobalBC);
}
